#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>     /* srand, rand */
#include <chrono>
#include <iostream>

__global__ void vectorAdd(int* a, int* b, int* c) 
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i] * (b[i] % 5);
    
    return;
}

template<typename Func>
void benchmark(Func func) {
    auto start = std::chrono::high_resolution_clock::now();
    
    func();
    
    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
    std::cout << "Time taken by function: " << duration.count() << " microseconds" << std::endl;
}

int main()
{
    const int items = 50000;
    int a[items];
    int b[items];

    for (int i = 0; i < items; i++) {
        a[i] = rand() % 1000000;
        b[i] = rand() % 1000000;
    }

    int c[sizeof(a) / sizeof(int)] = { 0 };
    int d[sizeof(a) / sizeof(int)] = { 0 };

    //create pointers into the gpu
    int* cudaA = 0;
    int* cudaB = 0;
    int* cudaC = 0;

    //allocate memory in the gpu
    hipMalloc(&cudaA, sizeof(a));
    hipMalloc(&cudaB, sizeof(b));
    hipMalloc(&cudaC, sizeof(c));

    //copy the vecors into the gpu
    hipMemcpy(cudaA, a, sizeof(a), hipMemcpyHostToDevice);
    hipMemcpy(cudaB, b, sizeof(b), hipMemcpyHostToDevice);


    benchmark([&]() {
        for (int i = 0; i < items; i++) {
            d[i] = a[i] + b[i] * (b[i] % 5);
        }
    });
    benchmark([&]() {
        vectorAdd <<<1, 1024 >> > (cudaA, cudaB, cudaC);
    });
        hipMemcpy(c, cudaC, sizeof(c), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    return 0;
}
